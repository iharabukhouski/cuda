#include "hip/hip_runtime.h"
#include "stdio.h"

__global__
void saxpy(
    int n,
    float a,
    float *x, // pointer
    float *y // pointer
) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n) {

        y[i] = a * x[i] + y[i];
    }

    // return;
}

int main(void) {

    int N = 1 << 20;

    float *x, *y, *d_x, *d_y;

    x = (float *)malloc(N * sizeof(float));
    y = (float *)malloc(N * sizeof(float));

    hipMalloc(&d_x, N * sizeof(float)); // hipMalloc - is from CUDA Runtime API
    hipMalloc(&d_y, N * sizeof(float));

    for (int i = 0; i < N; i++) {

        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);

    saxpy<<<(N + 255) / 256, 256>>>(N, 2.0f, d_x, d_y); // <<<number_of_thread_blocks, number_of_treads_per_block>>> - execution configuration

    hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);

    float maxError = 0.0f;
    for (int i = 0; i < N; i++) {

        maxError = max(maxError, abs(y[i] - 4.0f));
    }

    printf("Max error: %f\n", maxError);

    hipFree(d_x);
    hipFree(d_y);
    free(x);
    free(y);
}
